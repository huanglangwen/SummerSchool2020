#include "hip/hip_runtime.h"
#include "linalg.h"
#include "operators.h"

using data::Field;

template <typename F>
bool run_test(F f, const char* name) {
    auto success = f();
    printf("%-25s : ", name);
    if(!success) {
        printf("\033[1;31mfailed\033[0m\n");
        return false;
    }
    printf("\033[1;32mpassed\033[0m\n");
    return true;
} 
template <typename T>
bool check_value(T value, T expected, T tol) {
    if(std::fabs(value-expected)>tol) {
        std::cout << "  expected " << expected << " got " << value << std::endl;
        return false;
    }
    return true;
}

bool test_scaled_diff() {
    auto n = 5;
    Field y(n,1);
    Field l(n,1);
    Field r(n,1);

    for(auto i=0; i<n; ++i) {
        l[i] = 7.0;
        r[i] = 2.0;
    }
    l.update_device();
    r.update_device();

    linalg::ss_scaled_diff(y, 2.0, l, r);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], 10.0, 1.e-13);
    }
    return status;
}

bool test_fill() {
    auto n = 5;
    Field x(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
    }
    x.update_device();

    linalg::ss_fill(x, 2.0);
    x.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(x[i], 2.0, 1.e-13);
    }
    return status;
}

bool test_axpy() {
    auto n = 5;
    Field x(n,1);
    Field y(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
        y[i] = 5.0;
    }
    x.update_device();
    y.update_device();

    linalg::ss_axpy(y, 0.5, x);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], (0.5*3.0 + 5.0), 1.e-13);
    }
    return status;
}

bool test_add_scaled_diff() {
    auto n = 5;
    Field y(n,1);
    Field x(n,1);
    Field l(n,1);
    Field r(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
        l[i] = 7.0;
        r[i] = 2.0;
    }
    x.update_device();
    l.update_device();
    r.update_device();

    linalg::ss_add_scaled_diff(y, x, 1.5, l, r);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], 3. + 1.5 * (7. - 2.), 1.e-13);
    }
    return status;
}

bool test_scale() {
    auto n = 5;
    Field x(n,1);
    Field y(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
    }
    x.update_device();

    linalg::ss_scale(y, 0.5, x);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], 1.5, 1.e-13);
    }
    return status;
}

bool test_lcomb() {
    auto n = 5;
    Field x(n,1);
    Field y(n,1);
    Field z(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
        z[i] = 7.0;
    }
    x.update_device();
    z.update_device();

    linalg::ss_lcomb(y, 0.5, x, 2.0, z);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], (0.5*3. + 2.*7.), 1.e-13);
    }
    return status;
}

bool test_copy() {
    auto n = 5;
    Field x(n,1);
    Field y(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
    }

    x.update_device();
    linalg::ss_copy(y, x);
    y.update_host();

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y[i], x[i], 1.e-13);
    }

    return status;
}

bool test_dot() {
    auto n = 5;
    Field x(n,1);
    Field y(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
        y[i] = 7.0;
    }
    x.update_device();
    y.update_device();

    auto result = linalg::ss_dot(x, y);

    return check_value(result, n*3.*7., 1.e-13);
}

bool test_norm2() {
    auto n = 5;
    Field x(n,1);

    for(auto i=0; i<n; ++i) {
        x[i] = 2.0;
    }
    x.update_device();

    auto result = linalg::ss_norm2(x);

    return check_value(result, sqrt(2.0 * 2.0 * 5.0), 1.e-13);
}

void diffusion_cpu(const data::Field& U, data::Field& S, data::Discretization options)
{

    using data::bndE;
    using data::bndW;
    using data::bndN;
    using data::bndS;

    using data::x_old;

    double dxs = 1000. * (options.dx * options.dx);
    double alpha = options.alpha;
    int nx = options.nx;
    int ny = options.ny;
    int iend = nx - 1;
    int jend = ny - 1;

    // the interior grid points
#pragma omp parallel for
    for (int j = 1; j < jend; j++) {
        for (int i = 1; i < iend; i++) {
            S(i, j) = -(4. + alpha) * U(i, j)               // central point
                + U(i - 1, j) + U(i + 1, j) // east and west
                + U(i, j - 1) + U(i, j + 1) // north and south
                + alpha * x_old(i, j)
                + dxs * U(i, j) * (1.0 - U(i, j));
        }
    }

    // the east boundary
    {
        int i = nx - 1;
        for (int j = 1; j < jend; j++)
        {
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i - 1, j) + U(i, j - 1) + U(i, j + 1)
                + alpha * x_old(i, j) + bndE[j]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }
    }

    // the west boundary
    {
        int i = 0;
        for (int j = 1; j < jend; j++)
        {
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i + 1, j) + U(i, j - 1) + U(i, j + 1)
                + alpha * x_old(i, j) + bndW[j]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }
    }

    // the north boundary (plus NE and NW corners)
    {
        int j = ny - 1;

        {
            int i = 0; // NW corner
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i + 1, j) + U(i, j - 1)
                + alpha * x_old(i, j) + bndW[j] + bndN[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }

        // north boundary
        for (int i = 1; i < iend; i++)
        {
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i - 1, j) + U(i + 1, j) + U(i, j - 1)
                + alpha * x_old(i, j) + bndN[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }

        {
            int i = nx - 1; // NE corner
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i - 1, j) + U(i, j - 1)
                + alpha * x_old(i, j) + bndE[j] + bndN[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }
    }

    // the south boundary
    {
        int j = 0;

        {
            int i = 0; // SW corner
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i + 1, j) + U(i, j + 1)
                + alpha * x_old(i, j) + bndW[j] + bndS[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }

        // south boundary
        for (int i = 1; i < iend; i++)
        {
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i - 1, j) + U(i + 1, j) + U(i, j + 1)
                + alpha * x_old(i, j) + bndS[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }

        {
            int i = nx - 1; // SE corner
            S(i, j) = -(4. + alpha) * U(i, j)
                + U(i - 1, j) + U(i, j + 1)
                + alpha * x_old(i, j) + bndE[j] + bndS[i]
                + dxs * U(i, j) * (1.0 - U(i, j));
        }
    }

}

bool test_diffusion() {
    using data::options;
    using namespace data;
    auto nx = 128;
    auto ny = 128;
    auto t = 0.01;
    options.nx = nx;
    options.ny = ny;
    options.N = nx * ny;
    options.nt = 100;
    options.dt = t / options.nt;
    options.dx = 1. / (options.nx - 1);
    options.alpha = (options.dx * options.dx) / (1. * options.dt);
    Field x_new(nx, ny), x_new2(nx, ny);
    Field b(nx, ny), b2(nx,ny);

    x_new.init(nx, ny);
    x_old.init(nx, ny);
    bndN.init(nx, 1);
    bndS.init(nx, 1);
    bndE.init(ny, 1);
    bndW.init(ny, 1);

    linalg::ss_fill(bndN, 0.);
    linalg::ss_fill(bndS, 0.);
    linalg::ss_fill(bndE, 0.);
    linalg::ss_fill(bndW, 0.);

    linalg::ss_fill(x_new, 0.); x_new.update_host();
    linalg::ss_fill(b, 0.); 
    linalg::ss_fill(b2, 0.); b2.update_host();
    double xc = 1.0 / 4.0;
    double yc = (ny - 1) * options.dx / 4;
    double radius = fmin(xc, yc) / 2.0;
    for (int j = 0; j < ny; j++)
    {
        double y = (j - 1) * options.dx;
        for (int i = 0; i < nx; i++)
        {
            double x = (i - 1) * options.dx;
            if ((x - xc) * (x - xc) + (y - yc) * (y - yc) < radius * radius)
                x_new[i + nx * j] = 0.1;
        }
    }
    x_new.update_device();
    linalg::ss_copy(x_new2, x_new);
    x_new2.update_host();
    operators::diffusion(x_new, b); hipDeviceSynchronize();
    b.update_host();
    diffusion_cpu(x_new2, b2, options);

    double (*cmp1)[128][128], (*cmp2)[128][128];
    cmp1 = (decltype(cmp1))(b.host_data());
    cmp2 = (decltype(cmp2))(b2.host_data());
    
    bool status = true;
    for (auto i = 0; i < nx; ++i) {
        for (auto j = 0; j < ny; ++j) {
            status = status && check_value(*cmp1[i][j], *cmp2[i][j], 1.e-13);
        }   
    }

    return status;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(void) {
    run_test(test_dot,          "ss_dot");
    run_test(test_norm2,        "ss_norm2");
    run_test(test_scaled_diff,  "ss_scaled_diff");
    run_test(test_fill,         "ss_fill");
    run_test(test_axpy,         "ss_axpy");
    run_test(test_add_scaled_diff, "ss_add_scaled_diff");
    run_test(test_scale,        "ss_scale");
    run_test(test_lcomb,        "ss_lcomb");
    run_test(test_copy,         "ss_copy");
    run_test(test_diffusion,    "diffusion");
}

