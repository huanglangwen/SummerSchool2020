#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "util.hpp"

// TODO : implement a kernel that reverses a string of length n in place
__global__ void reverse_string_kernel(char* str, int n) {
    auto i = threadIdx.x;
    if (i * 2 < n) {
        auto tmp = str[i];
        str[i] = str[n - i - 1];
        str[n - i - 1] = tmp;
    }
}

__global__ void reverse_string2(char* str, int n) {
    auto i = threadIdx.x;
    if (i < n) {
        char buffer = str[n - i - 1];
        __syncthreads();
        str[i] = buffer;
    }
}

__host__ void reverse_string(char* str, int n) {
    reverse_string_kernel <<<1, n / 2 >>> (str, n);
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // TODO : call the string reverse function
    //reverse_string(string, n);
    reverse_string2<<<1, n>>>(string, n);

    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

